#include "hip/hip_runtime.h"
#include "tc.cuh"

void static_tc(GraphVine *graph, thrust::device_vector<float> &d_triangleCount) {

    auto &profiler = Profiler::getInstance();

    std::cout << "Page rank algorithm" << std::endl;

    // Get the device vertex dictionary
    VertexDictionary *device_vertex_dictionary = graph->getVertexDictionary();

    float *d_triangleCount_pointer = thrust::raw_pointer_cast(d_triangleCount.data());

    size_t vertex_size = graph->getVertexSize();
    size_t thread_blocks = ceil(double(vertex_size) / THREADS_PER_BLOCK);

    profiler.start("TC");
  
    tc_kernel_VC_sorted<<<thread_blocks, THREADS_PER_BLOCK>>>(device_vertex_dictionary, vertex_size, d_triangleCount_pointer);

    hipDeviceSynchronize();

    profiler.stop("TC");

    printDeviceVector("Triangle Count", d_triangleCount);

}

void dynamic_tc(GraphVine *graph, thrust::device_vector<float> &d_triangleCount){
    auto &profiler = Profiler::getInstance();

    // Get the device vertex dictionary
    VertexDictionary *device_vertex_dictionary = graph->getVertexDictionary();

    float *d_triangleCount_pointer = thrust::raw_pointer_cast(d_triangleCount.data());

    unsigned long *d_csr_offset_new_pointer = graph->getCsrOffsetNewPointer();
    unsigned long *d_csr_edges_new_pointer = graph->getCsrEdgesNewPointer();

    unsigned long *d_affected_nodes_pointer = graph->getAffectedNodesPointer();

    size_t vertex_size = graph->getVertexSize();
    size_t thread_blocks = ceil(double(vertex_size) / THREADS_PER_BLOCK);


    profiler.start("Dynamic Triangle Count");

    std::cout << "Starting dynamic triangle count algorithm" << std::endl;

    dynamic_tc_kernel_VC<<<thread_blocks, THREADS_PER_BLOCK>>>(device_vertex_dictionary, vertex_size, d_triangleCount_pointer, d_affected_nodes_pointer);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
    }

    hipDeviceSynchronize();

    profiler.stop("Dynamic Triangle Count");

    std::cout << "Stopping dynamic triangle count algorithm" << std::endl;

    printDeviceVector("Triangle Count", d_triangleCount);
}
