#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>

#include <chrono>

#include "marketReader.cuh"
#include "profiler.cuh"
#include "graphvine.cuh"
#include "csr.cuh"
#include "pagerank.cuh"
#include "tc.cuh"
#include "gpu.cuh"

size_t BATCH_SIZE = 10000000;

int main(int argc, char **argv){
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <market_file>" << " <GPU_ID>" << " <BATCH_SIZE>" << std::endl;
        return EXIT_FAILURE;
    }

    int gpuId = atoi(argv[2]);
    if(gpuId == -1){
        gpuId = selectLeastLoadedGPU();
    }

    BATCH_SIZE = atoi(argv[3]);

    // set the device to gpuId
    std::cout << "\n[+] Setting the GPU " << gpuId << "\n";
    CUDA_CALL(hipSetDevice(gpuId));
    // Print statistics regarding GPU DEVICE
    printDeviceStatistics(gpuId);

    auto &profiler = Profiler::getInstance();

    char *fileLoc = argv[1];
    CSR *csr = new CSR();
    csr->readGraph(fileLoc);


    GraphVine *dynGraph = new GraphVine(csr);

    // Give options to run benchmarks and applications
    // Insertion type operations
    size_t kk = 1;

    size_t vertex_size = CSR::h_graph_prop->xDim;

    // std::cout << "Starting Page Rank algorithm in main" << std::endl;

    // size_t free_mem, total_mem;
    // hipMemGetInfo(&free_mem, &total_mem);
    // std::cout << "Free memory: " << free_mem << ", Total memory: " << total_mem << std::endl;
    // std::cout << "Required memory: " << vertex_size * sizeof(float) * 2 << std::endl; // For two vectors

    // thrust::device_vector<float> d_pageRankVector_1(vertex_size);
    // thrust::device_vector<float> d_pageRankVector_2(vertex_size);

    // thrust::device_vector<float> d_triangleCount(vertex_size);

    // std::cout << "Starting Page Rank algorithm in main" << std::endl;

    // Run PR
    // static_pagerank(dynGraph, 0.85, 0.0001, 100, d_pageRankVector_1, d_pageRankVector_2);

    // Run TC
    // static_tc(dynGraph, d_triangleCount);

    int choice;
    std::cout << "Enter type of insertion required" << std::endl
              << "1. Regular batched edge insertion" << std::endl
              << "2. Edge Insert and Delete performance benchmark" << std::endl
              << "3. Vertex Insert and Delete performance benchmark" << std::endl;
    // std::cin >> choice;

    choice = 2;

    switch (choice) {
        case 1:
            // Regular batched edge insertion
            // dynGraph->bulkBuild();
            break;
        case 2:
            // Edge Insert and Delete performance benchmark
            profiler.start("Generate Random Batch");
            csr->generate_random_batch(BATCH_SIZE, kk);
            profiler.stop("Generate Random Batch");

            // dynGraph->batchInsert(csr, kk);

            // dynGraph->printDeviceVertexDictionary();

            dynGraph->batchDelete(csr, kk);

            // dynGraph->printDeviceVertexDictionary();

            // dynamic_pagerank(dynGraph, 0.85, 0.0001, 100, d_pageRankVector_1, d_pageRankVector_2);

            // static_pagerank(dynGraph, 0.85, 0.0001, 100, d_pageRankVector_1, d_pageRankVector_2);

            // dynamic_tc(dynGraph, d_triangleCount);

            // static_tc(dynGraph, d_triangleCount);

            break;
        case 3:
            // Vertex Insert and Delete performance benchmark
            break;
        default:
            std::cerr << "Invalid choice" << std::endl;
            break;
    }

    // Run Page rank algorithm
    

    return 0;
}