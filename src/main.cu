#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>

#include <chrono>

#include "marketReader.cuh"
#include "profiler.cuh"
#include "graphvine.cuh"
#include "csr.cuh"
#include "pagerank.cuh"


int main(int argc, char **argv){
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <market_file>" << " <GPU_ID>" << std::endl;
        return EXIT_FAILURE;
    }

    auto &profiler = Profiler::getInstance();

    char *fileLoc = argv[1];
    CSR *csr = new CSR();
    csr->readGraph(fileLoc);


    GraphVine *dynGraph = new GraphVine(csr);

    // Give options to run benchmarks and applications
    // Insertion type operations
    size_t kk = 0;

    size_t vertex_size = CSR::h_graph_prop->xDim;

    std::cout << "Starting Page Rank algorithm in main" << std::endl;

    thrust::device_vector<float> d_pageRankVector_1(vertex_size);
    thrust::device_vector<float> d_pageRankVector_2(vertex_size);

    // Run PR
    static_pagerank(dynGraph, 0.85, 0.0001, 100, d_pageRankVector_1, d_pageRankVector_2);

    int choice;
    std::cout << "Enter type of insertion required" << std::endl
              << "1. Regular batched edge insertion" << std::endl
              << "2. Edge Insert and Delete performance benchmark" << std::endl
              << "3. Vertex Insert and Delete performance benchmark" << std::endl;
    std::cin >> choice;

    switch (choice) {
        case 1:
            // Regular batched edge insertion
            // dynGraph->bulkBuild();
            break;
        case 2:
            // Edge Insert and Delete performance benchmark
            profiler.start("Generate Random Batch");
            csr->generate_random_batch(BATCH_SIZE, kk);
            profiler.stop("Generate Random Batch");

            dynGraph->batchInsert(csr, kk);

            dynamic_pagerank(dynGraph, 0.85, 0.0001, 100, d_pageRankVector_1, d_pageRankVector_2);

            static_pagerank(dynGraph, 0.85, 0.0001, 100, d_pageRankVector_1, d_pageRankVector_2);

            break;
        case 3:
            // Vertex Insert and Delete performance benchmark
            break;
        default:
            std::cerr << "Invalid choice" << std::endl;
            break;
    }

    // Run Page rank algorithm
    

    return 0;
}